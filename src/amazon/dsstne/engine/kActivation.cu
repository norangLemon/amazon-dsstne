#include "hip/hip_runtime.h"
/*


   Copyright 2016  Amazon.com, Inc. or its affiliates. All Rights Reserved.

   Licensed under the Apache License, Version 2.0 (the "License"). You may not use this file except in compliance with the License. A copy of the License is located at

   http://aws.amazon.com/apache2.0/

   or in the "license" file accompanying this file. This file is distributed on an "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the License for the specific language governing permissions and limitations under the License.
 */

#include "kernels.h"

#include "GpuContext.h"
#include "GpuTypes.h"
#include "NNTypes.h"
#include <limits>

static __constant__ GpuData cData;

__device__ inline uint64_t llitoulli(int64_t l)
{
    uint64_t u;
    asm("mov.b64    %0, %1;" : "=l"(u) : "l"(l));
    return u;
}

__device__ inline int64_t ullitolli(uint64_t u)
{
    int64_t l;
    asm("mov.b64    %0, %1;" : "=l"(l) : "l"(u));
    return l;
}

__device__ inline float atomicMax(float* address, float val)
{
    int* address_as_i   = (int*) address;
    int old             = *address_as_i, assumed;
    do 
    {
        assumed         = old;
        old             = ::atomicCAS(address_as_i, assumed, __float_as_int(::fmaxf(val, __int_as_float(assumed))));
    } 
    while (assumed != old);
    return __int_as_float(old);
}

void SetKActivationGpuData()
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cData), &(getGpu()._data), sizeof(GpuData));     
    RTERROR(status, "hipMemcpyToSymbol: SetKernelsGpuData copy to cData failed");
}

void GetKActivationGpuData()
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&(getGpu()._data), cData, sizeof(GpuData));     
    RTERROR(status, "hipMemcpyToSymbol: SetKernelsGpuData copy From cData failed");
}

__global__ void
LAUNCH_BOUNDS()
kCalculateSigmoidActivation_kernel(NNFloat* pData, uint64_t size)
{
    uint64_t pos                = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {
        NNFloat a               = 1.0f / (1.0f + exp(-pData[pos]));
        pData[pos]              = a;
    }
}


void kCalculateSigmoidActivation(NNFloat* pData, uint64_t size)
{
    uint32_t blocks             = CalculateBlocks(size);
    kCalculateSigmoidActivation_kernel<<<blocks, getGpu()._threadsPerBlock>>>(pData, size);
    LAUNCHERROR("kCalculateSigmoidActivation_kernel");
}

__global__ void
LAUNCH_BOUNDS()
kCalculateTanhActivation_kernel(NNFloat* pData, uint64_t size)
{
    uint64_t pos                = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
        pData[pos]              = tanh(pData[pos]);
}

void kCalculateTanhActivation(NNFloat* pData, uint64_t size)
{
    uint32_t blocks             = CalculateBlocks(size);
    kCalculateTanhActivation_kernel<<<blocks, getGpu()._threadsPerBlock>>>(pData, size);
    LAUNCHERROR("kCalculateTanhActivation_kernel");
}

__global__ void
LAUNCH_BOUNDS()
kCalculateReluActivation_kernel(NNFloat* pData, uint64_t size)
{
    uint64_t pos                = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
        pData[pos]              = max(0.0f, pData[pos]);
}

void kCalculateReluActivation(NNFloat* pData, uint64_t size)
{
    uint32_t blocks             = CalculateBlocks(size);
    kCalculateReluActivation_kernel<<<blocks, getGpu()._threadsPerBlock>>>(pData, size);
    LAUNCHERROR("kCalculateReluActivation_kernel");
}

__global__ void
LAUNCH_BOUNDS()
kCalculateSoftMaxActivation_kernel(NNFloat* pData, uint32_t stride)
{
    __shared__ unsigned long long int sAccumulator;
    __shared__ NNFloat sMaxValue;

    if (threadIdx.x == 0)
    {
        sAccumulator            = 0;
        sMaxValue               = (NNFloat)-99999999.0f;
    }
    __syncthreads();
    

    // Move data pointer to proper row, calculate activations, and sum them up as well as find maxmum output
    pData                      += blockIdx.x * stride;
    uint32_t pos                = threadIdx.x;
    NNFloat maxValue            = (NNFloat)-9999999999.0;
    
    // Calculate max value to improve numerical stability (Theano does this so I'll assume it's a good idea)
    while (pos < stride)
    {
        NNFloat z               = pData[pos];
        maxValue                = max(z, maxValue);
        pos                    += blockDim.x;
    }
    
    // Reduce maxValue within and between warps
    uint32_t tgx                = threadIdx.x & cData._warpMask;    
    maxValue                    = max(maxValue, __shfl(maxValue, tgx ^ 1));
    maxValue                    = max(maxValue, __shfl(maxValue, tgx ^ 2));
    maxValue                    = max(maxValue, __shfl(maxValue, tgx ^ 4));
    maxValue                    = max(maxValue, __shfl(maxValue, tgx ^ 8));
    maxValue                    = max(maxValue, __shfl(maxValue, tgx ^ 16));

    // Convert to 64-bit int to work around GPU instruction set deficiency
    if (tgx == 0) 
        atomicMax(&sMaxValue, maxValue);
    __syncthreads();        
    maxValue                    = sMaxValue;       

    // Calculate sum
    pos                         = threadIdx.x;
    NNFloat sum                 = (NNFloat)0.0;
    while (pos < stride)
    {
        NNFloat z               = pData[pos];
        sum                    += exp(z - maxValue);
        pos                    += blockDim.x;
    }    
         
    // Reduce sums within and between warps
    sum                        += __shfl(sum, tgx ^ 1);
    sum                        += __shfl(sum, tgx ^ 2);
    sum                        += __shfl(sum, tgx ^ 4);
    sum                        += __shfl(sum, tgx ^ 8);
    sum                        += __shfl(sum, tgx ^ 16);
    unsigned long long int lsum = llitoulli(llrintf(ERRORSCALEF * sum));
    if (tgx == 0) 
        atomicAdd(&sAccumulator, lsum);
    __syncthreads();               
    NNFloat norm                = (NNFloat)1.0 / (NNFloat)((double)sAccumulator * ONEOVERERRORSCALE);
    

    // Normalize output by dividing by sum of activations
    pos                         = threadIdx.x;
    while (pos < stride)
    {
        NNFloat z               = pData[pos];
        NNFloat a               = exp(z - maxValue);
        pData[pos]              = min((NNFloat)1.0, a * norm);
        pos                    += blockDim.x;
    }    

}
void kCalculateSoftMaxActivation(NNFloat* pData, uint32_t batch, uint32_t stride)
{
    uint32_t warps              = getGpu()._threadsPerBlock / getGpu()._warpSize;
    kCalculateSoftMaxActivation_kernel<<<batch, getGpu()._threadsPerBlock>>>(pData, stride);
    LAUNCHERROR("kCalculateSoftMaxActivation_kernel");
}


